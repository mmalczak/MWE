#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>

namespace matrix {

__global__ static void multiply_kernel(float *C, float *A, float *B, int m, int p, int n) {
  int i = threadIdx.y;
  int j = threadIdx.x;
  int k;
  float s = 0;
  for (k = 0; k < p; k++) {
    s += A[i * p + k] * B[k * n + j];
  }
  C[i * n + j] = s;
}

void multiply(float *C, float *A, float *B, int M, int P, int N) {
  float *devA, *devB, *devC;
  checkCudaErrors(hipSetDevice(0));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&devA), M * P * sizeof(float)));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&devB), P * N * sizeof(float)));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&devC), M * N * sizeof(float)));

  checkCudaErrors(hipMemcpy(devA, A, M * P * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(devB, B, P * N * sizeof(float), hipMemcpyHostToDevice));

  dim3 dimBlock(N, M);

  matrix::multiply_kernel<<<1, dimBlock>>>(devC, devA, devB, M, P, N);

  checkCudaErrors(hipGetLastError());

  checkCudaErrors(hipDeviceSynchronize());

  checkCudaErrors(hipMemcpy(C, devC, M * N * sizeof(float), hipMemcpyDeviceToHost));

  checkCudaErrors(hipFree(devA));
  checkCudaErrors(hipFree(devB));
  checkCudaErrors(hipFree(devC));
}

} // namespace matrix
